#include "hip/hip_runtime.h"

#include <gputk.h>

#define gpuTKCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      gpuTKLog(ERROR, "Failed to run stmt ", #stmt);                         \
      gpuTKLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int width, int numBCol) {
  //@@ Insert code to implement matrix multiplication here
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  if((Row < width) && (Col << width)){
    float Pvalue = 0;
    for(int k = 0; k < width; k++){
      Pvalue += A[Row * width + k] * B[k * numBCol + Col];
    }
    C[Row * numBCol + Col] = Pvalue;
  }

}

int main(int argc, char **argv) {
  gpuTKArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A3
  int numAColumns; // number of columns in the matrix A4
  int numBRows;    // number of rows in the matrix B4
  int numBColumns; // number of columns in the matrix B5
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)
  int width;
  int BLOCK_WIDTH = 4;

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)gpuTKImport(gpuTKArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows    = numARows;
  numCColumns = numBColumns;
  //@@ Set width of the matrix
  width = numARows;
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc((numCRows * numCColumns) * sizeof(float));

  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  gpuTKLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  gpuTKTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **) &deviceA, (numARows * numAColumns) * sizeof(float));
  hipMalloc((void **) &deviceB, (numBRows * numBColumns) * sizeof(float));
  hipMalloc((void **) &deviceC, (numCRows * numCColumns) * sizeof(float));
  gpuTKTime_stop(GPU, "Allocating GPU memory.");

  gpuTKTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, (numARows * numAColumns) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, (numBRows * numBColumns) * sizeof(float), hipMemcpyHostToDevice);
  gpuTKTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  // dim3 dimGrid(ceil((1.0*numCRows)/BLOCK_WIDTH), 
	// 		 ceil((1.0*numCRows)/BLOCK_WIDTH), 1);
  //dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);
  dim3 dimGrid(ceil((1.0*width)/BLOCK_WIDTH),ceil((1.0*width)/BLOCK_WIDTH),1);
  dim3 dimBlock(BLOCK_WIDTH,BLOCK_WIDTH,1);

  gpuTKTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiply<<<dimGrid,dimBlock>>>(deviceA, deviceB, deviceC, width, numBColumns);
  hipDeviceSynchronize();
  gpuTKTime_stop(Compute, "Performing CUDA computation");

  gpuTKTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here

  gpuTKTime_stop(Copy, "Copying output memory to the CPU");

  gpuTKTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here

  gpuTKTime_stop(GPU, "Freeing GPU Memory");

  gpuTKSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
